#include "hip/hip_runtime.h"
#include <stdio.h>
#define M 512
#define HIP_PI_F 3.141592654f

// the CUDA kernel for vector sum
__global__ void sum(double *a, double *b,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = a[idx]+b[idx];
	}
}

// the CUDA kernel for vector subtract
__global__ void subtract(double *a, double *b,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = a[idx]-b[idx];
	}
}

// the CUDA kernel for vector multiply
__global__ void multi(double *a, double *b,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = a[idx]*b[idx];
	}
}

// the CUDA kernel for vector divide
__global__ void divide(double *a, double *b,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] =  __ddiv_rn(a[idx],b[idx]);
	}
}

// the CUDA kernel for vector exp
__global__ void cudaexp(double *a,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = exp(a[idx]);
	}
}

// the CUDA kernel for vector log
__global__ void cudalog(double *a,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = log(a[idx]);
	}
}

// the CUDA kernel for vector square root
__global__ void cudasqrt(double *a,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] =  sqrt(a[idx]);
	}
}

// the CUDA kernel for gamma
__global__ void cudagamma(double *a,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] =  tgamma(a[idx]);
	}
}

// the CUDA kernel for beta
__global__ void cudabeta(double *a, double *b,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] =  tgamma(a[idx])*tgamma(b[idx])/tgamma(a[idx]+b[idx]);
	}
}


// the CUDA kernel for vector power
__global__ void cudapower(double *a,
	double *out, int n, double alpha)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = pow(a[idx], alpha);
	}
}

// the CUDA kernel for normal pdf
__global__ void cudanormdensity(double *a,
	double *out, int n, double mean, double sd)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = (1/(sd*sqrt(2*HIP_PI_F)))*exp(-pow((a[idx]-mean),2)/(2*pow(sd, 2)));
	}
}

// the CUDA kernel for normal CDF
__global__ void cudanormCDF(double *a,
	double *out, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = normcdf(a[idx]);
	}
}
	

//the CUDA kernel for sample variance
 __global__ void cuda_var(double *input, double *out, int n, double mean)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = pow(input[idx]-mean, 2);
	}

}


// the kernel for sample sum
__global__  void cudareduction(double * input, double * output, int len) 
{
	// Load a segment of the input vector into shared memory
	__shared__ double partialSum[2*M];
	int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int t = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;

	if ((start + t) < len)
	{
		partialSum[t] = input[start + t];      
	}
	else
	{       
		partialSum[t] = 0.0;
	}
	if ((start + blockDim.x + t) < len)
	{   
		partialSum[blockDim.x + t] = input[start + blockDim.x + t];
	}
	else
	{
		partialSum[blockDim.x + t] = 0.0;
	}

	// Traverse reduction tree
	for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
	{
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t + stride];
	}
	__syncthreads();

	// Write the computed sum of the block to the output vector at correct index
	if (t == 0 && (globalThreadId*2) < len)
	{
		output[blockIdx.x] = partialSum[t];
	}
}

// the CUDA kernel for vector subset copying
__global__ void vectorsubset(double *a, double *out, int n, int *index)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		out[idx] = a[index[idx]];
	}
}




extern "C"  void cuda_sum (double *a, double *b, double *c, int n)
{
	sum<<<(n+M-1)/M,M>>>(a, b, c, n);
	return;
}

extern "C"  void cuda_subtract (double *a, double *b, double *c, int n)
{
	subtract<<<(n+M-1)/M,M>>>(a, b, c, n);
	return;
}

extern "C"  void cuda_multi (double *a, double *b, double *c, int n)
{
	multi<<<(n+M-1)/M,M>>>(a, b, c, n);
	return;
}

extern "C"  void cuda_divide (double *a, double *b, double *c, int n)
{
	divide<<<(n+M-1)/M,M>>>(a, b, c, n);
	return;
}

extern "C"  void cuda_exp (double *a, double *c, int n)
{
	cudaexp<<<(n+M-1)/M,M>>>(a, c, n);
	return;
}

extern "C"  void cuda_log (double *a, double *c, int n)
{
	cudalog<<<(n+M-1)/M,M>>>(a, c, n);
	return;
}

extern "C"  void cuda_sqrt (double *a, double *c, int n)
{
	cudasqrt<<<(n+M-1)/M,M>>>(a, c, n);
	return;
}

extern "C"  void cuda_gamma (double *a, double *c, int n)
{
	cudagamma<<<(n+M-1)/M,M>>>(a, c, n);
	return;
}


extern "C"  void cuda_beta (double *a, double *b, double *c, int n)
{
	cudabeta<<<(n+M-1)/M,M>>>(a, b, c, n);
	return;
}



extern "C"  void cuda_power (double *a, double *c, int n, double alpha)
{
	cudapower<<<(n+M-1)/M,M>>>(a, c, n, alpha);
	return;
}

extern "C"  void cuda_normal_density(double *a, double *c, int n, double mean, double sd)
{
	cudanormdensity<<<(n+M-1)/M,M>>>(a, c, n, mean, sd);
	return;
}

extern "C"  void cuda_normal_CDF(double *a, double *c, int n)
{
	cudanormCDF<<<(n+M-1)/M,M>>>(a, c, n);
	return;
}


extern "C"  void cudavariance(double *a, double *c, int n, double mean)
{
	cuda_var<<<(n+M-1)/M,M>>>(a, c, n, mean);
	return;
}

extern "C"  void vector_subset (double *a, double *c, int n, int *index)
{
	vectorsubset<<<(n+M-1)/M,M>>>(a, c, n, index);
	return;
}


extern "C" double cuda_reduction (double *a, int n)
{

	int numOutputElements = n / (M<<1);
	if (n % (M<<1)) 
	{
		numOutputElements++;
	}
	double * hostOutput = (double*) malloc(numOutputElements * sizeof(double));
	double * deviceOutput;
	hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(double));
	dim3 DimGrid( numOutputElements, 1, 1);
	dim3 DimBlock(M, 1, 1);
	cudareduction<<<DimGrid, DimBlock>>>(a, deviceOutput, n);  
	hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(double), 
		hipMemcpyDeviceToHost);
	for (int ii = 1; ii < numOutputElements; ii++) 
	{
		hostOutput[0] += hostOutput[ii];
	}
	hipFree(deviceOutput); 
	return hostOutput[0];
}







// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
	printf("Major revision number:         %d\n",  devProp.major);
	printf("Minor revision number:         %d\n",  devProp.minor);
	printf("Name:                          %s\n",  devProp.name);
	printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
	printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
	printf("Warp size:                     %d\n",  devProp.warpSize);
	printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
	printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
	printf("Clock rate:                    %d\n",  devProp.clockRate);
	printf("Total constant memory:         %u\n",  devProp.totalConstMem);
	printf("Texture alignment:             %u\n",  devProp.textureAlignment);
	printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? 
		"Yes" : "No"));
	printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
	printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? 
		"Yes" : "No"));

}

extern "C" void gpuquery()
{
	// Number of CUDA devices
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("CUDA Device Query...\n");
	printf("There are %d CUDA devices.\n", devCount);

	// Iterate through devices
	for (int i = 0; i < devCount; ++i)
	{
		// Get device properties
		printf("\nCUDA Device #%d\n", i);
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		printDevProp(devProp);
	}

}


